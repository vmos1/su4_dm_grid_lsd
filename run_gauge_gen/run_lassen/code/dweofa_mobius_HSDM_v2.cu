#include "hip/hip_runtime.h"
/*************************************************************************************

Grid physics library, www.github.com/paboyle/Grid

Source file: ./tests/Test_hmc_EODWFRatio.cc

Copyright (C) 2015-2016

Author: Peter Boyle <pabobyle@ph.ed.ac.uk>
Author: Guido Cossu <guido.cossu@ed.ac.uk>

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation; either version 2 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License along
with this program; if not, write to the Free Software Foundation, Inc.,
51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.

See the full license in the file "LICENSE" in the top level distribution
directory
*************************************************************************************/
/*  END LEGAL */
#include <Grid/Grid.h>

namespace Grid{
  struct FermionParameters: Serializable {
    GRID_SERIALIZABLE_CLASS_MEMBERS(FermionParameters,
				    int, Ls,
				    double, mass,
				    double, M5,
				    double, b,
				    double, c,
				    double, StoppingCondition,
				    int, MaxCGIterations,
				    bool, ApplySmearing);

    //template <class ReaderClass >
    //FermionParameters(Reader<ReaderClass>& Reader){
    //  read(Reader, "Mobius", *this);
    //}

  };

  
  struct MobiusHMCParameters: Serializable {
  GRID_SERIALIZABLE_CLASS_MEMBERS(MobiusHMCParameters,
				  double, gauge_beta,
				  FermionParameters, Mobius)

  template <class ReaderClass >
  MobiusHMCParameters(Reader<ReaderClass>& Reader){
    read(Reader, "Action", *this);
  }

};

  struct SmearingParameters: Serializable {
    GRID_SERIALIZABLE_CLASS_MEMBERS(SmearingParameters,
				    double, rho,
				    Integer, Nsmear)

    template <class ReaderClass >
    SmearingParameters(Reader<ReaderClass>& Reader){
      read(Reader, "StoutSmearing", *this);
    }

  };
  
  
}


int main(int argc, char **argv) {
  using namespace Grid;
   ;

  Grid_init(&argc, &argv);
  int threads = GridThread::GetThreads();
  // here make a routine to print all the relevant information on the run
  std::cout << GridLogMessage << "Grid is setup to use " << threads << " threads" << std::endl;

   // Typedefs to simplify notation
  typedef GenericHMCRunner<MinimumNorm2> HMCWrapper;  // Uses the default minimum norm
  typedef WilsonImplR FermionImplPolicy;
  typedef MobiusFermionD FermionAction;
  typedef typename FermionAction::FermionField FermionField;
  // Serialiser
  typedef Grid::XmlReader       Serialiser;
  
  //::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::::
  HMCWrapper TheHMC;
  TheHMC.ReadCommandLine(argc, argv); // these can be parameters from file
 
  // Reader, file should come from command line
  if (TheHMC.ParameterFile.empty()){
    std::cout << "Input file not specified."
              << "Use --ParameterFile option in the command line.\nAborting" 
              << std::endl;
    exit(1);
  }
  Serialiser Reader(TheHMC.ParameterFile);

  MobiusHMCParameters MyParams(Reader);  
  // Apply smearing to the fermionic action
  bool ApplySmearing = MyParams.Mobius.ApplySmearing;
  
  

  // Grid from the command line
  TheHMC.Resources.AddFourDimGrid("gauge");
  // Possibile to create the module by hand 
  // hardcoding parameters or using a Reader

  // EOFA parameters
  OneFlavourRationalParams OFRp;
  OFRp.lo       = 0.98;
  OFRp.hi       = 25.0;
  OFRp.MaxIter  = 10000;
  OFRp.tolerance= 1.0e-7;
  OFRp.degree   = 10;
  OFRp.precision= 40;
  
  // Checkpointer definition (Name: Checkpointer)
  CheckpointerParameters CPparams(Reader);
  
  TheHMC.Resources.LoadNerscCheckpointer(CPparams);
  //  TheHMC.Resources.LoadBinaryCheckpointer(CPparams);

  // RNG definition (Name: RandomNumberGenerator)
  RNGModuleParameters RNGpar(Reader);
  TheHMC.Resources.SetRNGSeeds(RNGpar);

  // Construct observables
  // Plaquette and Polyakov loop
  typedef PlaquetteMod<HMCWrapper::ImplPolicy> PlaqObs;
  TheHMC.Resources.AddObservable<PlaqObs>();
    
  typedef PolyakovMod<HMCWrapper::ImplPolicy> PolyakovObs;
  TheHMC.Resources.AddObservable<PolyakovObs>();

  /////////////////////////////////////////////////////////////
  // Collect actions, here use more encapsulation
  // need wrappers of the fermionic classes 
  // that have a complex construction
  // standard

  WilsonGaugeActionR Waction(MyParams.gauge_beta);
//   SymanzikGaugeActionR Syzaction(MyParams.gauge_beta);
    
  const int Ls   = MyParams.Mobius.Ls;
  auto GridPtr   = TheHMC.Resources.GetCartesian();
  auto GridRBPtr = TheHMC.Resources.GetRBCartesian();
  auto FGrid     = SpaceTimeGrid::makeFiveDimGrid(Ls,GridPtr);
  auto FrbGrid   = SpaceTimeGrid::makeFiveDimRedBlackGrid(Ls,GridPtr);

  // temporarily need a gauge field
  LatticeGaugeField U(GridPtr);

  Real mass = MyParams.Mobius.mass; //0.04;
  Real pv   = 1.0;
  RealD M5  = MyParams.Mobius.M5; //1.5;
  RealD b   = MyParams.Mobius.b; //  3./2.;
  RealD c   = MyParams.Mobius.c; //  1./2.;

  // These lines are unecessary if BC are all periodic
  //std::vector<Complex> boundary = {1,1,1,-1};
  //FermionAction::ImplParams Params(boundary);
  

  ConjugateGradient<FermionField>  CG(MyParams.Mobius.StoppingCondition,MyParams.Mobius.MaxCGIterations);
  // DJM: setup for EOFA ratio (Mobius)
  MobiusEOFAFermionD Strange_Op_L(U, *FGrid, *FrbGrid, *GridPtr, *GridRBPtr, mass,     mass, pv,  0.0, -1, M5, b, c);
  MobiusEOFAFermionD Strange_Op_R(U, *FGrid, *FrbGrid, *GridPtr, *GridRBPtr, pv, mass, pv, -1.0,  1, M5, b, c);
  ExactOneFlavourRatioPseudoFermionAction<FermionImplPolicy> EOFA(Strange_Op_L, Strange_Op_R, CG, OFRp, true);
    
//   FermionAction DenOp(U,*FGrid,*FrbGrid,*GridPtr,*GridRBPtr,mass,M5,b,c, Params);
//   FermionAction NumOp(U,*FGrid,*FrbGrid,*GridPtr,*GridRBPtr,pv,  M5,b,c, Params);
//   TwoFlavourEvenOddRatioPseudoFermionAction<FermionImplPolicy> Nf2a(NumOp, DenOp,CG,CG);

  // Set smearing (true/false), default: false
  EOFA.is_smeared = ApplySmearing;
  
  // Collect actions
  ActionLevel<HMCWrapper::Field> Level1(1);
  Level1.push_back(&EOFA);


  ActionLevel<HMCWrapper::Field> Level2(4);
  Level2.push_back(&Waction);
//   Level2.push_back(&Syzaction);

  TheHMC.TheAction.push_back(Level1);
  TheHMC.TheAction.push_back(Level2);

  /////////////////////////////////////////////////////////////
  // HMC parameters are serialisable
  TheHMC.Parameters.initialize(Reader);

  // Reset performance counters 

  if (ApplySmearing){
    SmearingParameters SmPar(Reader);
    //double rho = 0.1;  // smearing parameter
    //int Nsmear = 3;    // number of smearing levels
    Smear_Stout<HMCWrapper::ImplPolicy> Stout(SmPar.rho);
    SmearedConfiguration<HMCWrapper::ImplPolicy> SmearingPolicy(GridPtr, SmPar.Nsmear, Stout);
    TheHMC.Run(SmearingPolicy); // for smearing
  } else {
    TheHMC.Run();  // no smearing
  }


  Grid_finalize();
} 

